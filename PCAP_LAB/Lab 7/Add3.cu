#include<stdio.h>
#include<string.h>
#include<hip/hip_runtime.h>


__global__ void format(char *in, int len,char *out,int *p)
{
  int idx=blockIdx.x*blockDim.x + threadIdx.x;

  if(idx<len)
  {
    int pos=atomicAdd(p,idx+1);

    for(int i=0;i<idx+1;i++)
    {
      out[pos+i]=in[idx];
    }
    printf("Thread %d: Appended %c at position %d for %d times\n", idx, in[idx], pos,idx+1);
  }
}

int main()
{
  char Sin[50],T[50];
  printf("Enter a string: ");
  scanf("%s",Sin);
  int len=strlen(Sin);
  int outlen=len*(len+1);

  char *d_out,*d_in;
  int *p;

  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start,0);
  hipMalloc((void **)&d_in,len);
  hipMalloc((void **)&d_out,outlen);
  hipMalloc((void **)&p,sizeof(int));

  hipMemcpy(d_in,Sin,len,hipMemcpyHostToDevice);
  hipMemset(p,0,sizeof(int));
  hipMemset(d_out,0,outlen);

  format<<<1,len>>>(d_in,len,d_out,p);

  hipError_t error=hipGetLastError();
  if(error!=hipSuccess)
  {
    printf("CUDA Error: %s\n",hipGetErrorString(error));
  }
  
  hipEventRecord(stop,0);
  float lapsed;
  hipEventElapsedTime(&lapsed,start,stop);

  hipMemcpy(T,d_out,outlen,hipMemcpyDeviceToHost);
  printf("Final Result: %s\n",T);
  printf("Time Lapsed: %f\n",lapsed);

  hipFree(d_in);
  hipFree(d_out);
  hipFree(p);

  return 0;
}
