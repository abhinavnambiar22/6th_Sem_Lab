#include<stdio.h>
#include<string.h>
#include<hip/hip_runtime.h>


__global__ void conc(char *in,int len,int N, char *out,int *p)
{
  int idx=blockIdx.x*blockDim.x + threadIdx.x;

  if(idx<N)
  {
    int pos=atomicAdd(p,len);

    for(int i=0;i<len;i++)
    {
      out[pos+i]=in[i];
    }
    printf("Thread %d: Wrote %s at position %d\n", idx, in, pos);
  }
}

int main()
{
  char Sin[50],Sout[50];
  printf("Enter a string: ");
  scanf("%s",Sin);

  int N;
  printf("Enter replication factor: ");
  scanf("%d",&N);

  int len=strlen(Sin);
  char *d_out,*d_in;
  int *p;

  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start,0);
  hipMalloc((void **)&d_in,len);
  hipMalloc((void **)&d_out,N*len+1);
  hipMalloc((void **)&p,sizeof(int));

  hipMemcpy(d_in,Sin,len,hipMemcpyHostToDevice);
  hipMemset(p,0,sizeof(int));
  hipMemset(d_out,0,N*len+1);

  conc<<<1,N>>>(d_in,len,N,d_out,p);

  hipError_t error=hipGetLastError();
  if(error!=hipSuccess)
  {
    printf("CUDA Error: %s\n",hipGetErrorString(error));
  }
  
  hipEventRecord(stop,0);
  float lapsed;
  hipEventElapsedTime(&lapsed,start,stop);

  hipMemcpy(Sout,d_out,N*len+1,hipMemcpyDeviceToHost);
  printf("Final Result: %s\n",Sout);
  printf("Time Lapsed: %f\n",lapsed);

  hipFree(d_in);
  hipFree(d_out);
  hipFree(p);

  return 0;
}
